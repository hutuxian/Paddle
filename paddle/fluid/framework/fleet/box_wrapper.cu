
#include <hip/hip_runtime.h>
// Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#ifdef PADDLE_WITH_BOX_PS
#include <algorithm>
#include <ctime>
#include <memory>
#include <numeric>
#include "paddle/fluid/framework/fleet/box_wrapper.h"
#include "paddle/fluid/framework/lod_tensor.h"
#include "paddle/fluid/platform/gpu_info.h"

namespace paddle {
namespace framework {
#define CUDA_KERNEL_LOOP(i, n)                                 \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)

template <size_t EMBEDX_DIM, size_t EXPAND_EMBED_DIM>
__global__ void PullCopy(
    float** dest,
    const boxps::FeatureValueGpu<EMBEDX_DIM, EXPAND_EMBED_DIM>* src,
    const int64_t* len, int hidden, int expand_dim, int slot_num, int total_len,
    uint64_t** keys) {
  CUDA_KERNEL_LOOP(i, total_len) {
    int low = 0;
    int high = slot_num - 1;
    while (low < high) {
      int mid = (low + high) / 2;
      if (i < len[mid])
        high = mid;
      else
        low = mid + 1;
    }
    int x = low;
    int y = i - (x ? len[x - 1] : 0);
    if (*(keys[x] + y) == 0) {
      *(dest[x] + y * hidden) = 0;
      *(dest[x] + y * hidden + 1) = 0;
      *(dest[x] + y * hidden + 2) = 0;
    } else {
      *(dest[x] + y * hidden) = (src + i)->show;
      *(dest[x] + y * hidden + 1) = (src + i)->clk;
      *(dest[x] + y * hidden + 2) = (src + i)->embed_w;
    }
    if ((src + i)->embedding_size == 0 || *(keys[x] + y) == 0) {
      for (int j = 0; j < hidden - 3; j++) {
        *(dest[x] + y * hidden + 3 + j) = 0;
      }
    } else {
      for (int j = 0; j < hidden - 3; j++) {
        *(dest[x] + y * hidden + 3 + j) = (src + i)->embedx[1 + j];
      }
    }
    // process embed_expand
    if (expand_dim > 0) {
      int z = x + slot_num;
      if ((src + i)->embed_expand_size[0] == 0 || *(keys[x] + y) == 0) {
        for (int j = 0; j < expand_dim; j++) {
          *(dest[z] + y * expand_dim + j) = 0;
        }
      } else {
        for (int j = 0; j < expand_dim; j++) {
          *(dest[z] + y * expand_dim + j) = (src + i)->embed_expand[1 + j];
        }
      }
    }
  }  // end kernel loop
}

__global__ void CopyKeysKernel(uint64_t** src_keys, uint64_t* dest_total_keys,
                               const int64_t* len, int slot_num,
                               int total_len) {
  CUDA_KERNEL_LOOP(i, total_len) {
    int low = 0;
    int high = slot_num - 1;
    while (low < high) {
      int mid = (low + high) / 2;
      if (i < len[mid])
        high = mid;
      else
        low = mid + 1;
    }
    int x = low;
    int y = i - (x ? len[x - 1] : 0);
    dest_total_keys[i] = src_keys[x][y];
  }
}

template <size_t EMBEDX_DIM, size_t EXPAND_EMBED_DIM>
__global__ void PushCopy(
    boxps::FeaturePushValueGpu<EMBEDX_DIM, EXPAND_EMBED_DIM>* dest, float** src,
    int64_t* len, int hidden, int expand_dim, int slot_num, int total_len,
    int bs, int* slot_vector) {
  CUDA_KERNEL_LOOP(i, total_len) {
    int low = 0;
    int high = slot_num - 1;
    while (low < high) {
      int mid = (low + high) / 2;
      if (i < len[mid])
        high = mid;
      else
        low = mid + 1;
    }
    int x = low;
    int y = i - (x ? len[low - 1] : 0);
    (dest + i)->slot = slot_vector[x];
    (dest + i)->show = *(src[x] + y * hidden);
    (dest + i)->clk = *(src[x] + y * hidden + 1);
    (dest + i)->embed_g = *(src[x] + y * hidden + 2) * -1. * bs;
    for (int j = 0; j < hidden - 3; j++) {
      (dest + i)->embedx_g[j] = *(src[x] + y * hidden + 3 + j) * -1. * bs;
    }
    if (expand_dim > 0) {
      int z = x + slot_num;
      for (int j = 0; j < expand_dim; j++) {
        (dest + i)->embed_expand_g[j] =
            *(src[z] + y * expand_dim + j) * -1. * bs;
      }
    }
  }
}

__global__ void AddBasicCalculator(const float* pred, const int64_t* label,
                                   double* positive, double* negative,
                                   double* abs_error, double* sqr_error,
                                   double* local_pred, int len,
                                   int table_size) {
  CUDA_KERNEL_LOOP(ins_idx, len) {
    int pos = static_cast<int>(pred[ins_idx] * table_size);
    if (pos >= table_size) {
      pos = table_size - 1;
    }
    if (label[ins_idx] == 0) {
      atomicAdd(negative + pos, 1.0);
      // negative[pos]++;
    } else {
      atomicAdd(positive + pos, 1.0);
      // positive[pos]++;
    }
    double err = pred[ins_idx] - label[ins_idx];
    abs_error[ins_idx] += fabs(err);
    sqr_error[ins_idx] += err * err;
    local_pred[ins_idx] += pred[ins_idx];
  }
}

void BoxWrapper::CopyForPull(const paddle::platform::Place& place,
                             uint64_t** gpu_keys,
                             const std::vector<float*>& values,
                             void* total_values_gpu, const int64_t* gpu_len,
                             const int slot_num, const int hidden_size,
                             const int expand_embed_dim,
                             const int64_t total_length) {
  auto stream = dynamic_cast<platform::CUDADeviceContext*>(
                    platform::DeviceContextPool::Instance().Get(
                        BOOST_GET_CONST(platform::CUDAPlace, place)))
                    ->stream();
  auto buf_value = memory::AllocShared(place, values.size() * sizeof(float*));
  float** gpu_values = reinterpret_cast<float**>(buf_value->ptr());
  cudaMemcpy(gpu_values, values.data(), values.size() * sizeof(float*),
             cudaMemcpyHostToDevice);
#define EMBEDX_CASE(i, ...)                                                  \
  case i: {                                                                  \
    constexpr size_t EmbedxDim = i;                                          \
    switch (expand_embed_dim) {                                              \
      __VA_ARGS__                                                            \
      default:                                                               \
        PADDLE_THROW(platform::errors::InvalidArgument(                      \
            "Unsupport this expand embedding size [%d]", expand_embed_dim)); \
    }                                                                        \
  } break

#define EXPAND_EMBED_PULL_CASE(i, ...)                                       \
  case i: {                                                                  \
    constexpr size_t ExpandDim = i;                                          \
    PullCopy<EmbedxDim,                                                      \
             ExpandDim><<<(total_length + 512 - 1) / 512, 512, 0, stream>>>( \
        gpu_values,                                                          \
        reinterpret_cast<boxps::FeatureValueGpu<EmbedxDim, ExpandDim>*>(     \
            total_values_gpu),                                               \
        gpu_len, hidden_size, expand_embed_dim, slot_num, total_length,      \
        gpu_keys);                                                           \
  } break

  switch (hidden_size - 3) {
    EMBEDX_CASE(8, EXPAND_EMBED_PULL_CASE(0); EXPAND_EMBED_PULL_CASE(8);
                EXPAND_EMBED_PULL_CASE(64););
    EMBEDX_CASE(16, EXPAND_EMBED_PULL_CASE(0););
    EMBEDX_CASE(256, EXPAND_EMBED_PULL_CASE(0););
    EMBEDX_CASE(128, EXPAND_EMBED_PULL_CASE(0););
    default:
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Unsupport this embedding size [%d]", hidden_size - 3));
  }
  cudaStreamSynchronize(stream);
#undef EXPAND_EMBED_PULL_CASE
#undef EMBEDX_CASE
}

void BoxWrapper::CopyKeys(const paddle::platform::Place& place,
                          uint64_t** origin_keys, uint64_t* total_keys,
                          const int64_t* gpu_len, int slot_num, int total_len) {
  auto stream = dynamic_cast<platform::CUDADeviceContext*>(
                    platform::DeviceContextPool::Instance().Get(
                        BOOST_GET_CONST(platform::CUDAPlace, place)))
                    ->stream();
  CopyKeysKernel<<<(total_len + 512 - 1) / 512, 512, 0, stream>>>(
      origin_keys, total_keys, gpu_len, slot_num, total_len);
  cudaStreamSynchronize(stream);
}

void BoxWrapper::CopyForPush(const paddle::platform::Place& place,
                             const std::vector<const float*>& grad_values,
                             void* total_grad_values_gpu,
                             const std::vector<int64_t>& slot_lengths,
                             const int hidden_size, const int expand_embed_dim,
                             const int64_t total_length, const int batch_size) {
  auto stream = dynamic_cast<platform::CUDADeviceContext*>(
                    platform::DeviceContextPool::Instance().Get(
                        BOOST_GET_CONST(platform::CUDAPlace, place)))
                    ->stream();
  auto slot_lengths_lod = slot_lengths;
  for (int i = 1; i < slot_lengths_lod.size(); i++) {
    slot_lengths_lod[i] += slot_lengths_lod[i - 1];
  }
  auto buf_grad_value =
      memory::AllocShared(place, grad_values.size() * sizeof(float*));
  auto buf_length =
      memory::AllocShared(place, slot_lengths.size() * sizeof(int64_t));
  auto buf_slot_vector =
      memory::AllocShared(place, slot_lengths_lod.size() * sizeof(int));

  float** gpu_values = reinterpret_cast<float**>(buf_grad_value->ptr());
  int64_t* gpu_len = reinterpret_cast<int64_t*>(buf_length->ptr());
  int* d_slot_vector = reinterpret_cast<int*>(buf_slot_vector->ptr());

  cudaMemcpy(gpu_values, grad_values.data(),
             grad_values.size() * sizeof(float*), cudaMemcpyHostToDevice);
  cudaMemcpy(gpu_len, slot_lengths_lod.data(),
             slot_lengths.size() * sizeof(int64_t), cudaMemcpyHostToDevice);
  cudaMemcpy(d_slot_vector, slot_vector_.data(),
             slot_lengths_lod.size() * sizeof(int), cudaMemcpyHostToDevice);

#define EMBEDX_CASE(i, ...)                                                  \
  case i: {                                                                  \
    constexpr size_t EmbedxDim = i;                                          \
    switch (expand_embed_dim) {                                              \
      __VA_ARGS__                                                            \
      default:                                                               \
        PADDLE_THROW(platform::errors::InvalidArgument(                      \
            "Unsupport this expand embedding size [%d]", expand_embed_dim)); \
    }                                                                        \
  } break

#define EXPAND_EMBED_PUSH_CASE(i, ...)                                       \
  case i: {                                                                  \
    constexpr size_t ExpandDim = i;                                          \
    PushCopy<EmbedxDim,                                                      \
             ExpandDim><<<(total_length + 512 - 1) / 512, 512, 0, stream>>>( \
        reinterpret_cast<boxps::FeaturePushValueGpu<EmbedxDim, ExpandDim>*>( \
            total_grad_values_gpu),                                          \
        gpu_values, gpu_len, hidden_size, expand_embed_dim,                  \
        slot_lengths.size(), total_length, batch_size, d_slot_vector);       \
  } break

  switch (hidden_size - 3) {
    EMBEDX_CASE(8, EXPAND_EMBED_PUSH_CASE(0); EXPAND_EMBED_PUSH_CASE(8);
                EXPAND_EMBED_PUSH_CASE(64););
    EMBEDX_CASE(16, EXPAND_EMBED_PUSH_CASE(0););
    EMBEDX_CASE(256, EXPAND_EMBED_PUSH_CASE(0););
    EMBEDX_CASE(128, EXPAND_EMBED_PUSH_CASE(0););
    default:
      PADDLE_THROW(platform::errors::InvalidArgument(
          "Unsupport this embedding size [%d]", hidden_size - 3));
  }

  cudaStreamSynchronize(stream);
#undef EXPAND_EMBED_PUSH_CASE
#undef EMBEDX_CASE
}

void BasicAucCalculator::cuda_add_data(const paddle::platform::Place& place,
                                       const int64_t* label, const float* pred,
                                       int len) {

  auto stream = dynamic_cast<platform::CUDADeviceContext*>(
                    platform::DeviceContextPool::Instance().Get(
                        BOOST_GET_CONST(platform::CUDAPlace, place)))
                    ->stream();

  int i = BOOST_GET_CONST(platform::CUDAPlace, place).GetDeviceId();

  cudaSetDevice(i);

  AddBasicCalculator<<<(len + 512 - 1) / 512, 512, 0, stream>>>(
      pred, label, reinterpret_cast<double*>(_d_positive[i]->ptr()),
      reinterpret_cast<double*>(_d_negative[i]->ptr()),
      reinterpret_cast<double*>(_d_abserr[i]->ptr()),
      reinterpret_cast<double*>(_d_sqrerr[i]->ptr()),
      reinterpret_cast<double*>(_d_pred[i]->ptr()), len, _table_size);
}

__global__
void pull_query_emb_kernel(int len, int dim, uint64_t* key, float* val, float* table) {
    CUDA_KERNEL_LOOP(i, len) {
        val[i] = table[key[i / dim] * dim + i % dim];
    }
}

void QueryEmbSet::PullQueryEmb(uint64_t* d_keys, float* d_vals, int num, int gpu_id) {
  auto place = platform::CUDAPlace(gpu_id);
  auto stream = dynamic_cast<platform::CUDADeviceContext*>(
                    platform::DeviceContextPool::Instance().Get(place))
                    ->stream();
  int len = emb_dim * num;
  const int BLOCK_SIZE_ = 256;
  pull_query_emb_kernel<<<(len + BLOCK_SIZE_ - 1) / BLOCK_SIZE_, BLOCK_SIZE_, 0, stream>>>(len, emb_dim, d_keys, d_vals, d_embs[gpu_id]);
  //std::vector<float> h;
  //h.resize(128);
  
  //h_emb_mtx.lock();
  //std::cout << "val:";
  //cudaMemcpyAsync(h.data(), d_vals + 4 * 128, sizeof(float) * 128, cudaMemcpyDeviceToHost, stream);
  //cudaStreamSynchronize(stream);
  //for (int i = 0; i < 128; ++i) {
  //  std::cout << " " << h[i];
  //}
  //std::cout << std::endl;
  //h_emb_mtx.unlock();
}

}  // end namespace framework
}  // end namespace paddle
#endif
